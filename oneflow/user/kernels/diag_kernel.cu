#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/user/kernels/diag_kernel.h"

namespace oneflow {
namespace {

template<typename T>
__global__ void copy_to_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size,
                                        int32_t strideSum) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i * (strideSum)] = in_buf[i]; }
}

template<typename T>
__global__ void copy_from_diagonal_kernel(T* out_buf, const T* in_buf, int32_t size,
                                          int32_t strideSum) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i] = in_buf[i * (strideSum)]; }
}

template<typename T>
__global__ void copy_to_diagonal_grad_kernel(T* out_buf, const T* in_buf, int32_t size,
                                             int32_t strideSum) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i] = in_buf[i * (strideSum)]; }
}

template<typename T>
__global__ void copy_from_diagonal_grad_kernel(T* out_buf, const T* in_buf, int32_t size,
                                               int32_t strideSum) {
  CUDA_1D_KERNEL_LOOP(i, size) { out_buf[i * (strideSum)] = in_buf[i]; }
}

template<typename T>
struct DiagFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, T* out_buf, const T* in_buf, int32_t sz, int32_t strideSum,
                  int32_t in_dim) {
    if (in_dim == 1) {
      copy_to_diagonal_kernel<<<BlocksNum4ThreadsNum(sz * sz), kCudaThreadsNumPerBlock, 0,
                                ctx->hip_stream()>>>(out_buf, in_buf, sz, int32_t(strideSum));
    } else {
      copy_from_diagonal_kernel<<<BlocksNum4ThreadsNum(sz * sz), kCudaThreadsNumPerBlock, 0,
                                  ctx->hip_stream()>>>(out_buf, in_buf, sz, int32_t(strideSum));
    }
  }
};

template<typename T>
struct DiagGradFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, T* dx_buf, const T* dy_buf, int32_t dx_num_cnt,
                  int32_t dy_num_cnt, int32_t strideSum, int32_t in_dim) {
    if (in_dim == 1) {
      copy_to_diagonal_grad_kernel<<<BlocksNum4ThreadsNum(dx_num_cnt), kCudaThreadsNumPerBlock, 0,
                                     ctx->hip_stream()>>>(dx_buf, dy_buf, dx_num_cnt,
                                                           int32_t(strideSum));
    } else {
      copy_from_diagonal_grad_kernel<<<BlocksNum4ThreadsNum(dy_num_cnt), kCudaThreadsNumPerBlock, 0,
                                       ctx->hip_stream()>>>(dx_buf, dy_buf, dy_num_cnt,
                                                             int32_t(strideSum));
    }
  }
};

}  // namespace

REGISTER_DIAG_KERNELS(DeviceType::kGPU, half);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, float);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, double);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int8_t);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int32_t);
REGISTER_DIAG_KERNELS(DeviceType::kGPU, int64_t);

}  // namespace oneflow
#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

template<typename T>
struct GeluFunctor {
  OF_DEVICE_FUNC T compute(T x, int64_t i) const {
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x));
  }
};

template<typename T>
struct MaskAndScaleFunctor {
  MaskAndScaleFunctor(const int8_t* mask, T scale) : mask(mask), scale(scale) {}
  OF_DEVICE_FUNC T compute(T x, int64_t i) const { return x * static_cast<T>(mask[i]) * scale; }
  const int8_t* mask;
  T scale;
};

template<typename T>
struct MaskAndScaleAddFunctor {
  MaskAndScaleAddFunctor(const int8_t* mask, const T* addend, T scale)
      : mask(mask), addend(addend), scale(scale) {}
  OF_DEVICE_FUNC T compute(T x, int64_t i) const {
    return x * static_cast<T>(mask[i]) * scale + addend[i];
  }
  const int8_t* mask;
  const T* addend;
  T scale;
};

template<typename T>
struct GeluGradFunctor {
  const T coef = sqrt(static_cast<T>(2.0) / acos(static_cast<T>(-1.0)));
  OF_DEVICE_FUNC T compute(T x, T dy, int64_t i) const {
    return static_cast<T>(0.5)
           * (static_cast<T>(1.0) + erf(static_cast<T>(M_SQRT1_2) * x)
              + x * coef * exp(static_cast<T>(-0.5) * x * x))
           * dy;
  }
};

template<>
struct GeluFunctor<half> {
  GeluFunctor<float> float_functor;
  OF_DEVICE_FUNC half compute(half x, int64_t i) const {
    return __float2half(float_functor.compute(__half2float(x), i));
  }
};

template<>
struct GeluGradFunctor<half> {
  GeluGradFunctor<float> float_functor;
  OF_DEVICE_FUNC half compute(half x, half dy, int64_t i) const {
    return __float2half(float_functor.compute(__half2float(x), __half2float(dy), i));
  }
};

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                const Index inner_size, const T* x, const T* bias, T* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    y[i] = functor.compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                    const Index bias_size, const Index inner_size, const T* x,
                                    const T* bias, const T* dy, T* dx) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[(i % block_size) / inner_size];
    dx[i] = grad_functor.compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddGpuHalf(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                    const Index inner_size, const half* x, const half* bias,
                                    half* y) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    float x_i = __half2float(x[i]) + __half2float(bias[(i % block_size) / inner_size]);
    y[i] = __float2half(functor.compute(x_i, i));
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddGradGpuHalf(FUNCTOR grad_functor, const Index elem_cnt,
                                        const Index bias_size, const Index inner_size,
                                        const half* x, const half* bias, const half* dy, half* dx) {
  const Index block_size = bias_size * inner_size;
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    float x_i = __half2float(x[i]) + __half2float(bias[(i % block_size) / inner_size]);
    dx[i] = __float2half(grad_functor.compute(x_i, __half2float(dy[i]), i));
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddRowGpu(FUNCTOR functor, const Index elem_cnt, const Index bias_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    y[i] = functor.compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradRowGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index bias_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i % bias_size];
    dx[i] = grad_functor.compute(x_i, dy[i], i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddRowGpuHalf2(FUNCTOR functor, const Index elem_cnt,
                                        const Index bias_size, const half* x, const half* bias,
                                        half* y) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    float x_i_0 = __half2float(x_h2[i].x) + __half2float(bias_h2[i % h2_bias_size].x);
    float x_i_1 = __half2float(x_h2[i].y) + __half2float(bias_h2[i % h2_bias_size].y);
    float2 y_i;
    y_i.x = functor.compute(x_i_0, 2 * i);
    y_i.y = functor.compute(x_i_1, 2 * i + 1);
    y_h2[i] = __float22half2_rn(y_i);
  }
}

template<typename FUNCTOR, typename Index>
__global__ void FusedBiasAddGradRowGpuHalf2(FUNCTOR grad_functor, const Index elem_cnt,
                                            const Index bias_size, const half* x, const half* bias,
                                            const half* dy, half* dx) {
  const Index h2_elem_cnt = elem_cnt / 2;
  const Index h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  const auto* dy_h2 = reinterpret_cast<const half2*>(dy);
  auto* dx_h2 = reinterpret_cast<half2*>(dx);
  CUDA_1D_KERNEL_LOOP_T(Index, i, h2_elem_cnt) {
    float x_i_0 = __half2float(x_h2[i].x) + __half2float(bias_h2[i % h2_bias_size].x);
    float x_i_1 = __half2float(x_h2[i].y) + __half2float(bias_h2[i % h2_bias_size].y);
    float2 dy_i = __half2float2(dy_h2[i]);
    float2 dx_i;
    dx_i.x = grad_functor.compute(x_i_0, __half2float(dy_i.x), 2 * i);
    dx_i.y = grad_functor.compute(x_i_1, __half2float(dy_i.y), 2 * i + 1);
    dx_h2[i] = __float22half2_rn(dx_i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddColGpu(FUNCTOR functor, const Index elem_cnt, const Index inner_size,
                                   const T* x, const T* bias, T* y) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    y[i] = functor.compute(x_i, i);
  }
}

template<typename FUNCTOR, typename T, typename Index>
__global__ void FusedBiasAddGradColGpu(FUNCTOR grad_functor, const Index elem_cnt,
                                       const Index inner_size, const T* x, const T* bias,
                                       const T* dy, T* dx) {
  CUDA_1D_KERNEL_LOOP_T(Index, i, elem_cnt) {
    T x_i = x[i] + bias[i / inner_size];
    dx[i] = grad_functor.compute(x_i, dy[i], i);
  }
}

}  // namespace

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddCalculation {
  static void Invoke(DeviceCtx* ctx, FUNCTOR functor, Index outer_size, Index bias_size,
                     Index inner_size, const T* x, const T* bias, T* y) {
    const Index elem_cnt = outer_size * bias_size * inner_size;
    if (inner_size == 1) {
      FusedBiasAddRowGpu<FUNCTOR, T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              functor, elem_cnt, bias_size, x, bias, y);
    } else if (outer_size == 1) {
      FusedBiasAddColGpu<FUNCTOR, T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              functor, elem_cnt, inner_size, x, bias, y);
    } else {
      RUN_CUDA_KERNEL((FusedBiasAddGpu<FUNCTOR, T, Index>), ctx, elem_cnt, functor, elem_cnt,
                      bias_size, inner_size, x, bias, y);
    }
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddCalculation<FUNCTOR, float16, Index> {
  static void Invoke(DeviceCtx* ctx, FUNCTOR functor, Index outer_size, Index bias_size,
                     Index inner_size, const float16* x, const float16* bias, float16* y) {
    const Index elem_cnt = outer_size * bias_size * inner_size;
    if (inner_size == 1) {
      if (bias_size % 2 == 0) {
        FusedBiasAddRowGpuHalf2<FUNCTOR, Index><<<BlocksNum4ThreadsNum(elem_cnt / 2),
                                                  kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
            functor, elem_cnt, bias_size, reinterpret_cast<const half*>(x),
            reinterpret_cast<const half*>(bias), reinterpret_cast<half*>(y));
      } else {
        FusedBiasAddRowGpu<FUNCTOR, half, Index>
            <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
                functor, elem_cnt, bias_size, reinterpret_cast<const half*>(x),
                reinterpret_cast<const half*>(bias), reinterpret_cast<half*>(y));
      }
    } else if (outer_size == 1) {
      FusedBiasAddColGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              functor, elem_cnt, inner_size, reinterpret_cast<const half*>(x),
              reinterpret_cast<const half*>(bias), reinterpret_cast<half*>(y));
    } else {
      RUN_CUDA_KERNEL((FusedBiasAddGpuHalf<FUNCTOR, Index>), ctx, elem_cnt, functor, elem_cnt,
                      bias_size, inner_size, reinterpret_cast<const half*>(x),
                      reinterpret_cast<const half*>(bias), reinterpret_cast<half*>(y));
    }
  }
};

template<typename FUNCTOR, typename T, typename Index>
struct FusedBiasAddGradCalculation {
  static void Invoke(DeviceCtx* ctx, FUNCTOR grad_functor, Index outer_size, Index bias_size,
                     Index inner_size, const T* x, const T* bias, const T* dy, T* dx) {
    const Index elem_cnt = outer_size * bias_size * inner_size;
    if (inner_size == 1) {
      FusedBiasAddGradRowGpu<FUNCTOR, T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              grad_functor, elem_cnt, bias_size, x, bias, dy, dx);
    } else if (outer_size == 1) {
      FusedBiasAddGradColGpu<FUNCTOR, T, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              grad_functor, elem_cnt, inner_size, x, bias, dy, dx);
    } else {
      RUN_CUDA_KERNEL((FusedBiasAddGradGpu<FUNCTOR, T, Index>), ctx, elem_cnt, grad_functor,
                      elem_cnt, bias_size, inner_size, x, bias, dy, dx);
    }
  }
};

template<typename FUNCTOR, typename Index>
struct FusedBiasAddGradCalculation<FUNCTOR, float16, Index> {
  static void Invoke(DeviceCtx* ctx, FUNCTOR grad_functor, Index outer_size, Index bias_size,
                     Index inner_size, const float16* x, const float16* bias, const float16* dy,
                     float16* dx) {
    const Index elem_cnt = outer_size * bias_size * inner_size;
    if (inner_size == 1) {
      if (bias_size % 2 == 0) {
        FusedBiasAddGradRowGpuHalf2<FUNCTOR, Index>
            <<<BlocksNum4ThreadsNum(elem_cnt / 2), kCudaThreadsNumPerBlock, 0,
               ctx->hip_stream()>>>(
                grad_functor, elem_cnt, bias_size, reinterpret_cast<const half*>(x),
                reinterpret_cast<const half*>(bias), reinterpret_cast<const half*>(dy),
                reinterpret_cast<half*>(dx));
      } else {
        FusedBiasAddGradRowGpu<FUNCTOR, half, Index>
            <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
                grad_functor, elem_cnt, bias_size, reinterpret_cast<const half*>(x),
                reinterpret_cast<const half*>(bias), reinterpret_cast<const half*>(dy),
                reinterpret_cast<half*>(dx));
      }
    } else if (outer_size == 1) {
      FusedBiasAddGradColGpu<FUNCTOR, half, Index>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
              grad_functor, elem_cnt, inner_size, reinterpret_cast<const half*>(x),
              reinterpret_cast<const half*>(bias), reinterpret_cast<const half*>(dy),
              reinterpret_cast<half*>(dx));
    } else {
      RUN_CUDA_KERNEL((FusedBiasAddGradGpuHalf<FUNCTOR, Index>), ctx, elem_cnt, grad_functor,
                      elem_cnt, bias_size, inner_size, reinterpret_cast<const half*>(x),
                      reinterpret_cast<const half*>(bias), reinterpret_cast<const half*>(dy),
                      reinterpret_cast<half*>(dx));
    }
  }
};

template<typename T>
class FusedFusedBiasAddKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddKernel() = default;
  ~FusedFusedBiasAddKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    GeluFunctor<T> gelu_functor;
    if (IsKernelSafeInt32(n)) {
      FusedBiasAddCalculation<decltype(gelu_functor), T, int32_t>::Invoke(
          ctx->device_ctx(), gelu_functor, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
          b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    } else {
      FusedBiasAddCalculation<decltype(gelu_functor), T, int64_t>::Invoke(
          ctx->device_ctx(), gelu_functor, outer_size, bias_size, inner_size, a_tensor->dptr<T>(),
          b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(dtype)        \
  REGISTER_USER_KERNEL("fused_bias_add_gelu")             \
      .SetCreateFn<FusedFusedBiasAddKernel<dtype>>()      \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_KERNEL(half)

template<typename T>
class FusedBiasAddMaskScaleKernel final : public user_op::OpKernel {
 public:
  FusedBiasAddMaskScaleKernel() = default;
  ~FusedBiasAddMaskScaleKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* mask_tensor = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const float scale = ctx->Attr<float>("scale");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    if (ctx->user_op_conf().has_input("_add_to_output", 0)) {
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      MaskAndScaleAddFunctor<T> mask_and_scale_add_functor(mask_tensor->dptr<int8_t>(),
                                                           addend->dptr<T>(), scale);
      if (IsKernelSafeInt32(n)) {
        FusedBiasAddCalculation<decltype(mask_and_scale_add_functor), T, int32_t>::Invoke(
            ctx->device_ctx(), mask_and_scale_add_functor, outer_size, bias_size, inner_size,
            a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
      } else {
        FusedBiasAddCalculation<decltype(mask_and_scale_add_functor), T, int64_t>::Invoke(
            ctx->device_ctx(), mask_and_scale_add_functor, outer_size, bias_size, inner_size,
            a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
      }
    } else {
      MaskAndScaleFunctor<T> mask_and_scale_functor(mask_tensor->dptr<int8_t>(), scale);
      if (IsKernelSafeInt32(n)) {
        FusedBiasAddCalculation<decltype(mask_and_scale_functor), T, int32_t>::Invoke(
            ctx->device_ctx(), mask_and_scale_functor, outer_size, bias_size, inner_size,
            a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
      } else {
        FusedBiasAddCalculation<decltype(mask_and_scale_functor), T, int64_t>::Invoke(
            ctx->device_ctx(), mask_and_scale_functor, outer_size, bias_size, inner_size,
            a_tensor->dptr<T>(), b_tensor->dptr<T>(), out_tensor->mut_dptr<T>());
      }
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(dtype)  \
  REGISTER_USER_KERNEL("fused_bias_add_mask_scale")       \
      .SetCreateFn<FusedBiasAddMaskScaleKernel<dtype>>()  \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_MASK_SCALE_KERNEL(half)

template<typename T>
class FusedFusedBiasAddGradKernel final : public user_op::OpKernel {
 public:
  FusedFusedBiasAddGradKernel() = default;
  ~FusedFusedBiasAddGradKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* a_tensor = ctx->Tensor4ArgNameAndIndex("a", 0);
    const auto* b_tensor = ctx->Tensor4ArgNameAndIndex("b", 0);
    const auto* dy_tensor = ctx->Tensor4ArgNameAndIndex("dy", 0);
    auto* dx_tensor = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int32_t bias_add_axis = ctx->Attr<int32_t>("axis");
    const int64_t outer_size = a_tensor->shape().Count(0, bias_add_axis);
    const int64_t bias_size = a_tensor->shape().At(bias_add_axis);
    const int64_t inner_size = a_tensor->shape().Count(bias_add_axis + 1);
    const auto n = a_tensor->shape().elem_cnt();
    GeluGradFunctor<T> gelu_grad_functor;
    if (IsKernelSafeInt32(n)) {
      FusedBiasAddGradCalculation<decltype(gelu_grad_functor), T, int32_t>::Invoke(
          ctx->device_ctx(), gelu_grad_functor, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    } else {
      FusedBiasAddGradCalculation<decltype(gelu_grad_functor), T, int64_t>::Invoke(
          ctx->device_ctx(), gelu_grad_functor, outer_size, bias_size, inner_size,
          a_tensor->dptr<T>(), b_tensor->dptr<T>(), dy_tensor->dptr<T>(), dx_tensor->mut_dptr<T>());
    }
  };

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(dtype)   \
  REGISTER_USER_KERNEL("fused_bias_add_gelu_grad")        \
      .SetCreateFn<FusedFusedBiasAddGradKernel<dtype>>()  \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu") \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(float)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(double)
REGISTER_FUSED_BIAS_ADD_GELU_GRAD_KERNEL(half)

}  // namespace oneflow
